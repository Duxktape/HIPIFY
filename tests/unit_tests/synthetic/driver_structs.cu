// RUN: %run_test hipify "%s" "%t" %hipify_args -D__CUDA_API_VERSION_INTERNAL %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int main() {
  printf("01. CUDA Driver API Structs synthetic test\n");

  // CHECK: HIP_ARRAY3D_DESCRIPTOR array3D_st;
  // CHECK-NEXT: HIP_ARRAY3D_DESCRIPTOR array3D;
  // CHECK-NEXT: HIP_ARRAY3D_DESCRIPTOR array3D_v2;
  HIP_ARRAY3D_DESCRIPTOR array3D_st;
  HIP_ARRAY3D_DESCRIPTOR array3D;
  HIP_ARRAY3D_DESCRIPTOR array3D_v2;

  // CHECK: HIP_ARRAY_DESCRIPTOR array_descr_st;
  // CHECK-NEXT: HIP_ARRAY_DESCRIPTOR array_descr;
  HIP_ARRAY_DESCRIPTOR array_descr_st;
  HIP_ARRAY_DESCRIPTOR array_descr;
#define __CUDA_API_VERSION_INTERNAL
  // CHECK: HIP_ARRAY_DESCRIPTOR array_descr_v1_st;
  // CHECK-NEXT: HIP_ARRAY_DESCRIPTOR array_descr_v1;
  HIP_ARRAY_DESCRIPTOR array_descr_v1_st;
  HIP_ARRAY_DESCRIPTOR array_descr_v1;
#undef __CUDA_API_VERSION_INTERNAL
  // CHECK: HIP_ARRAY_DESCRIPTOR array_descr_v2;
  HIP_ARRAY_DESCRIPTOR array_descr_v2;

  // CHECK: hipExternalMemoryBufferDesc_st ext_mem_buff_st;
  // CHECK-NEXT: hipExternalMemoryBufferDesc ext_mem_buff;
  // CHECK-NEXT: hipExternalMemoryBufferDesc ext_mem_buff_v1;
  hipExternalMemoryBufferDesc_st ext_mem_buff_st;
  hipExternalMemoryBufferDesc ext_mem_buff;
  hipExternalMemoryBufferDesc ext_mem_buff_v1;

  // CHECK: hipExternalMemoryHandleDesc_st ext_mem_handle_st;
  // CHECK-NEXT: hipExternalMemoryHandleDesc ext_mem_handle;
  // CHECK-NEXT: hipExternalMemoryHandleDesc ext_mem_handle_v1;
  hipExternalMemoryHandleDesc_st ext_mem_handle_st;
  hipExternalMemoryHandleDesc ext_mem_handle;
  hipExternalMemoryHandleDesc ext_mem_handle_v1;

  // CHECK: hipExternalSemaphoreHandleDesc_st ext_sema_handle_st;
  // CHECK-NEXT: hipExternalSemaphoreHandleDesc ext_sema_handle;
  // CHECK-NEXT: hipExternalSemaphoreHandleDesc ext_sema_handle_v1;
  hipExternalSemaphoreHandleDesc_st ext_sema_handle_st;
  hipExternalSemaphoreHandleDesc ext_sema_handle;
  hipExternalSemaphoreHandleDesc ext_sema_handle_v1;

  // CHECK: hipExternalSemaphoreSignalParams_st ext_sema_params_st;
  // CHECK-NEXT: hipExternalSemaphoreSignalParams ext_sema_params;
  // CHECK-NEXT: hipExternalSemaphoreSignalParams ext_sema_params_v1;
  hipExternalSemaphoreSignalParams_st ext_sema_params_st;
  hipExternalSemaphoreSignalParams ext_sema_params;
  hipExternalSemaphoreSignalParams ext_sema_params_v1;

  // CHECK: hipHostNodeParams host_node_params_st;
  // CHECK-NEXT: hipHostNodeParams host_node_params;
  // CHECK-NEXT: hipHostNodeParams host_node_params_v1;
  hipHostNodeParams host_node_params_st;
  hipHostNodeParams host_node_params;
  hipHostNodeParams host_node_params_v1;

  // CHECK: hipKernelNodeParams kern_node_params_st;
  // CHECK-NEXT: hipKernelNodeParams kern_node_params;
  // CHECK-NEXT: hipKernelNodeParams kern_node_params_v1;
  hipKernelNodeParams kern_node_params_st;
  hipKernelNodeParams kern_node_params;
  hipKernelNodeParams kern_node_params_v1;

  // CHECK: hip_Memcpy2D memcpy2D_st;
  // CHECK-NEXT: hip_Memcpy2D memcpy2D_v1_st;
  // CHECK-NEXT: hip_Memcpy2D memcpy2D;
  // CHECK-NEXT: hip_Memcpy2D memcpy2D_v1;
  // CHECK-NEXT: hip_Memcpy2D memcpy2D_v2;
  hip_Memcpy2D memcpy2D_st;
  hip_Memcpy2D memcpy2D_v1_st;
  hip_Memcpy2D memcpy2D;
  hip_Memcpy2D memcpy2D_v1;
  hip_Memcpy2D memcpy2D_v2;

  // CHECK: HIP_MEMCPY3D memcpy3D_st;
  // CHECK-NEXT: HIP_MEMCPY3D memcpy3D_v1_st;
  // CHECK-NEXT: HIP_MEMCPY3D memcpy3D;
  // CHECK-NEXT: HIP_MEMCPY3D memcpy3D_v1;
  // CHECK-NEXT: HIP_MEMCPY3D memcpy3D_v2;
  HIP_MEMCPY3D memcpy3D_st;
  HIP_MEMCPY3D memcpy3D_v1_st;
  HIP_MEMCPY3D memcpy3D;
  HIP_MEMCPY3D memcpy3D_v1;
  HIP_MEMCPY3D memcpy3D_v2;

  // CHECK: HIP_RESOURCE_DESC_st res_descr_st;
  // CHECK-NEXT: HIP_RESOURCE_DESC res_descr;
  // CHECK-NEXT: HIP_RESOURCE_DESC res_descr_v1;
  HIP_RESOURCE_DESC_st res_descr_st;
  HIP_RESOURCE_DESC res_descr;
  HIP_RESOURCE_DESC res_descr_v1;

  // CHECK: HIP_RESOURCE_VIEW_DESC_st res_view_descr_st;
  // CHECK-NEXT: HIP_RESOURCE_VIEW_DESC res_view_descr;
  // CHECK-NEXT: HIP_RESOURCE_VIEW_DESC res_view_descr_v1;
  HIP_RESOURCE_VIEW_DESC_st res_view_descr_st;
  HIP_RESOURCE_VIEW_DESC res_view_descr;
  HIP_RESOURCE_VIEW_DESC res_view_descr_v1;

  // CHECK: HIP_TEXTURE_DESC_st tex_descr_st;
  // CHECK-NEXT: HIP_TEXTURE_DESC tex_descr;
  // CHECK-NEXT: HIP_TEXTURE_DESC tex_descr_v1;
  HIP_TEXTURE_DESC_st tex_descr_st;
  HIP_TEXTURE_DESC tex_descr;
  HIP_TEXTURE_DESC tex_descr_v1;

  // CHECK: hipIpcMemHandle_st ipc_mem_handle_st;
  // CHECK-NEXT: hipIpcMemHandle_t ipc_mem_handle;
  // CHECK-NEXT: hipIpcMemHandle_t ipc_mem_handle_v1;
  hipIpcMemHandle_st ipc_mem_handle_st;
  hipIpcMemHandle_t ipc_mem_handle;
  hipIpcMemHandle_t ipc_mem_handle_v1;

  // CHECK: hipArray* array_st_ptr;
  // CHECK-NEXT: hipArray* array_ptr;
  hipArray* array_st_ptr;
  hipArray_t array_ptr;

  // CHECK: ihipCtx_t* ctx_st_ptr;
  // CHECK-NEXT: hipCtx_t ctx;
  ihipCtx_t* ctx_st_ptr;
  hipCtx_t ctx;

  // CHECK: ihipEvent_t* evnt_st_ptr;
  // CHECK-NEXT: hipEvent_t evnt;
  ihipEvent_t* evnt_st_ptr;
  hipEvent_t evnt;

  // CHECK: hipExternalMemory_t ext_mem;
  hipExternalMemory_t ext_mem;

  // CHECK: hipExternalSemaphore_t ext_sema;
  hipExternalSemaphore_t ext_sema;

  // CHECK: ihipModuleSymbol_t* func_st_ptr;
  // CHECK-NEXT: hipFunction_t func;
  ihipModuleSymbol_t* func_st_ptr;
  hipFunction_t func;

  // CHECK: hipMipmappedArray* mipmapped_array_st_ptr;
  // CHECK-NEXT: hipMipmappedArray_t mipmapped_array;
  hipMipmappedArray* mipmapped_array_st_ptr;
  hipMipmappedArray_t mipmapped_array;

  // CHECK: ihipStream_t* stream_st_ptr;
  // CHECK-NEXT: hipStream_t stream;
  ihipStream_t* stream_st_ptr;
  hipStream_t stream;

  // CHECK: textureReference* tex_ref_st_ptr;
  // CHECK-NEXT: hipTexRef tex_ref;
  textureReference* tex_ref_st_ptr;
  hipTexRef tex_ref;

  // CHECK: hipGraph* graph_st;
  // CHECK-NEXT: hipGraph_t graph;
  ihipGraph* graph_st;
  hipGraph_t graph;

  // CHECK: hipGraphExec* graphExec_st;
  // CHECK-NEXT: hipGraphExec_t graphExec;
  hipGraphExec* graphExec_st;
  hipGraphExec_t graphExec;

  // CHECK: hipGraphicsResource* graphicsResource_st;
  // CHECK-NEXT: hipGraphicsResource_t graphicsResource;
  hipGraphicsResource* graphicsResource_st;
  hipGraphicsResource_t graphicsResource;

  return 0;
}
