// RUN: %run_test hipify "%s" "%t" %hipify_args -D__CUDA_API_VERSION_INTERNAL %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

int main() {
  printf("08. CUDA Runtime API Defines synthetic test\n");

  // CHECK: int IPC_HANDLE_SIZE = HIP_IPC_HANDLE_SIZE;
  // CHECK-NEXT: int ArrayDefault = hipArrayDefault;
  // CHECK-NEXT: int ArrayLayered = hipArrayLayered;
  // CHECK-NEXT: int ArraySurfaceLoadStore = hipArraySurfaceLoadStore;
  // CHECK-NEXT: int ArrayCubemap = hipArrayCubemap;
  // CHECK-NEXT: int ArrayTextureGather = hipArrayTextureGather;
  // CHECK-NEXT: int CooperativeLaunchMultiDeviceNoPreSync = hipCooperativeLaunchMultiDeviceNoPreSync;
  // CHECK-NEXT: int CooperativeLaunchMultiDeviceNoPostSync = hipCooperativeLaunchMultiDeviceNoPostSync;
  // CHECK-NEXT: int CpuDeviceId = hipCpuDeviceId;
  // CHECK-NEXT: int InvalidDeviceId = hipInvalidDeviceId;
  // CHECK-NEXT: int DeviceBlockingSync = hipDeviceScheduleBlockingSync;
  // CHECK-NEXT: int DeviceLmemResizeToMax = hipDeviceLmemResizeToMax;
  // CHECK-NEXT: int DeviceMapHost = hipDeviceMapHost;
  // CHECK-NEXT: int DeviceScheduleAuto = hipDeviceScheduleAuto;
  // CHECK-NEXT: int DeviceScheduleSpin = hipDeviceScheduleSpin;
  // CHECK-NEXT: int DeviceScheduleYield = hipDeviceScheduleYield;
  // CHECK-NEXT: int DeviceScheduleBlockingSync = hipDeviceScheduleBlockingSync;
  // CHECK-NEXT: int DeviceScheduleMask = hipDeviceScheduleMask;
  // CHECK-NEXT: int EventDefault = hipEventDefault;
  // CHECK-NEXT: int EventBlockingSync = hipEventBlockingSync;
  // CHECK-NEXT: int EventDisableTiming = hipEventDisableTiming;
  // CHECK-NEXT: int EventInterprocess = hipEventInterprocess;
  int IPC_HANDLE_SIZE = HIP_IPC_HANDLE_SIZE;
  int ArrayDefault = hipArrayDefault;
  int ArrayLayered = hipArrayLayered;
  int ArraySurfaceLoadStore = hipArraySurfaceLoadStore;
  int ArrayCubemap = hipArrayCubemap;
  int ArrayTextureGather = hipArrayTextureGather;
  int CooperativeLaunchMultiDeviceNoPreSync = hipCooperativeLaunchMultiDeviceNoPreSync;
  int CooperativeLaunchMultiDeviceNoPostSync = hipCooperativeLaunchMultiDeviceNoPostSync;
  int CpuDeviceId = hipCpuDeviceId;
  int InvalidDeviceId = hipInvalidDeviceId;
  int DeviceBlockingSync = hipDeviceScheduleBlockingSync;
  int DeviceLmemResizeToMax = hipDeviceLmemResizeToMax;
  int DeviceMapHost = hipDeviceMapHost;
  int DeviceScheduleAuto = hipDeviceScheduleAuto;
  int DeviceScheduleSpin = hipDeviceScheduleSpin;
  int DeviceScheduleYield = hipDeviceScheduleYield;
  int DeviceScheduleBlockingSync = hipDeviceScheduleBlockingSync;
  int DeviceScheduleMask = hipDeviceScheduleMask;
  int EventDefault = hipEventDefault;
  int EventBlockingSync = hipEventBlockingSync;
  int EventDisableTiming = hipEventDisableTiming;
  int EventInterprocess = hipEventInterprocess;

  return 0;
}
