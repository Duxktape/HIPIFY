// RUN: %run_test hipify "%s" "%t" %hipify_args -D__CUDA_API_VERSION_INTERNAL %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

int main() {
  printf("05. CUDA Runtime API Structs synthetic test\n");

  // CHECK: hipChannelFormatDesc ChannelFormatDesc;
  hipChannelFormatDesc ChannelFormatDesc;

  // CHECK: hipDeviceProp_t DeviceProp;
  hipDeviceProp_t DeviceProp;

  // CHECK: hipExtent Extent;
  hipExtent Extent;

  // CHECK: hipExternalMemoryBufferDesc ExternalMemoryBufferDesc;
  hipExternalMemoryBufferDesc ExternalMemoryBufferDesc;

  // CHECK: hipExternalMemoryHandleDesc ExternalMemoryHandleDesc;
  hipExternalMemoryHandleDesc ExternalMemoryHandleDesc;

  // CHECK: hipExternalSemaphoreHandleDesc ExternalSemaphoreHandleDesc;
  hipExternalSemaphoreHandleDesc ExternalSemaphoreHandleDesc;

  // CHECK: hipExternalSemaphoreSignalParams ExternalSemaphoreSignalParams;
  // CHECK-NEXT: hipExternalSemaphoreSignalParams ExternalSemaphoreSignalParams_v1;
  hipExternalSemaphoreSignalParams ExternalSemaphoreSignalParams;
  hipExternalSemaphoreSignalParams ExternalSemaphoreSignalParams_v1;

  // CHECK: hipExternalSemaphoreWaitParams ExternalSemaphoreWaitParams;
  // CHECK-NEXT: hipExternalSemaphoreWaitParams ExternalSemaphoreWaitParams_v1;
  hipExternalSemaphoreWaitParams ExternalSemaphoreWaitParams;
  hipExternalSemaphoreWaitParams ExternalSemaphoreWaitParams_v1;

  // CHECK: hipFuncAttributes FuncAttributes;
  hipFuncAttributes FuncAttributes;

  // CHECK: hipHostNodeParams HostNodeParams;
  hipHostNodeParams HostNodeParams;

  // CHECK: hipIpcEventHandle_st IpcEventHandle_st;
  // CHECK-NEXT: hipIpcEventHandle_t IpcEventHandle_t;
  hipIpcEventHandle_st IpcEventHandle_st;
  hipIpcEventHandle_t IpcEventHandle_t;

  // CHECK: hipIpcMemHandle_st IpcMemHandle_st;
  // CHECK-NEXT: hipIpcMemHandle_t IpcMemHandle_t;
  hipIpcMemHandle_st IpcMemHandle_st;
  hipIpcMemHandle_t IpcMemHandle_t;

  // CHECK: hipKernelNodeParams KernelNodeParams;
  hipKernelNodeParams KernelNodeParams;

  // CHECK: hipLaunchParams LaunchParams;
  hipLaunchParams LaunchParams;

  // CHECK: hipMemcpy3DParms Memcpy3DParms;
  hipMemcpy3DParms Memcpy3DParms;

  // CHECK: hipMemsetParams MemsetParams;
  hipMemsetParams MemsetParams;

  // CHECK: hipPitchedPtr PitchedPtr;
  hipPitchedPtr PitchedPtr;

  // CHECK: hipPointerAttribute_t PointerAttributes;
  hipPointerAttribute_t PointerAttributes;

  // CHECK: hipPos Pos;
  hipPos Pos;

  // CHECK: hipResourceDesc ResourceDesc;
  hipResourceDesc ResourceDesc;

  // CHECK: hipResourceViewDesc ResourceViewDesc;
  hipResourceViewDesc ResourceViewDesc;

  // CHECK: hipTextureDesc TextureDesc;
  hipTextureDesc TextureDesc;

  // CHECK: surfaceReference surfaceRef;
  surfaceReference surfaceRef;

  // CHECK: ihipEvent_t* event_st;
  // CHECK-NEXT: hipEvent_t Event_t;
  ihipEvent_t* event_st;
  hipEvent_t Event_t;

  // CHECK: hipExternalMemory_t ExternalMemory_t;
  hipExternalMemory_t ExternalMemory_t;

  // CHECK: hipExternalSemaphore_t ExternalSemaphore_t;
  hipExternalSemaphore_t ExternalSemaphore_t;

  // CHECK: hipGraph* graph_st;
  // CHECK-NEXT: hipGraph_t Graph_t;
  ihipGraph* graph_st;
  hipGraph_t Graph_t;

  // CHECK: hipGraphExec* graphExec_st;
  // CHECK-NEXT: hipGraphExec_t GraphExec_t;
  hipGraphExec* graphExec_st;
  hipGraphExec_t GraphExec_t;

  // CHECK: hipGraphicsResource* GraphicsResource;
  // CHECK-NEXT: hipGraphicsResource_t GraphicsResource_t;
  hipGraphicsResource* GraphicsResource;
  hipGraphicsResource_t GraphicsResource_t;

  // CHECK: hipGraphNode* graphNode_st;
  // CHECK-NEXT: hipGraphNode_t GraphNode_t;
  hipGraphNode* graphNode_st;
  hipGraphNode_t GraphNode_t;

  // CHECK: hipArray* Array;
  // CHECK-NEXT: hipArray_t Array_t;
  // CHECK-NEXT: hipArray_const_t Array_const_t;
  hipArray* Array;
  hipArray_t Array_t;
  hipArray_const_t Array_const_t;

  // CHECK: hipMipmappedArray* MipmappedArray;
  // CHECK-NEXT: hipMipmappedArray_t MipmappedArray_t;
  // CHECK-NEXT: hipMipmappedArray_const_t MipmappedArray_const_t;
  hipMipmappedArray* MipmappedArray;
  hipMipmappedArray_t MipmappedArray_t;
  hipMipmappedArray_const_t MipmappedArray_const_t;

  // CHECK: ihipStream_t* stream_st;
  // CHECK-NEXT: hipStream_t Stream_t;
  ihipStream_t* stream_st;
  hipStream_t Stream_t;

  // CHECK: ihipModuleSymbol_t* func_st_ptr;
  // CHECK-NEXT: hipFunction_t func;
  ihipModuleSymbol_t* func_st_ptr;
  hipFunction_t func;

  return 0;
}
