// RUN: %run_test hipify "%s" "%t" %hipify_args -D__CUDA_API_VERSION_INTERNAL %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int main() {
  printf("03. CUDA Driver API Typedefs synthetic test\n");

  // CHECK: hipDevice_t device;
  // CHECK-NEXT: hipDevice_t device_v1;
  hipDevice_t device;
  hipDevice_t device_v1;

  // CHECK: hipDeviceptr_t deviceptr;
  // CHECK-NEXT: hipDeviceptr_t deviceptr_v1;
  // CHECK-NEXT: hipDeviceptr_t deviceptr_v2;
  hipDeviceptr_t deviceptr;
  hipDeviceptr_t deviceptr_v1;
  hipDeviceptr_t deviceptr_v2;

  // CHECK: hipHostFn_t hostFn;
  hipHostFn_t hostFn;

  // CHECK: hipStreamCallback_t streamCallback;
  hipStreamCallback_t streamCallback;

  // CHECK: hipSurfaceObject_t surfObject;
  // CHECK-NEXT: hipSurfaceObject_t surfObject_v1;
  hipSurfaceObject_t surfObject;
  hipSurfaceObject_t surfObject_v1;

  // CHECK: hipTextureObject_t texObject;
  // CHECK-NEXT: hipTextureObject_t texObject_v1;
  hipTextureObject_t texObject;
  hipTextureObject_t texObject_v1;

  return 0;
}
