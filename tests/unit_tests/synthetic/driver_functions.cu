// RUN: %run_test hipify "%s" "%t" %hipify_args 1 --skip-excluded-preprocessor-conditional-blocks %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <stdio.h>

int main() {
  printf("09. CUDA Driver API Functions synthetic test\n");

  unsigned int flags = 0;
  size_t bytes = 0;
  size_t bytes_2 = 0;
  void* image = nullptr;
  std::string name = "str";
  // CHECK: hipDevice_t device;
  // CHECK-NEXT: hipCtx_t context;
  // CHECK-NEXT: hipFuncCache_t func_cache;
  // CHECK-NEXT: hipLimit_t limit;
  // CHECK-NEXT: hipSharedMemConfig pconfig;
  // CHECK-NEXT: hipFunction_t function;
  // CHECK-NEXT: hipModule_t module_;
  // CHECK-NEXT: hipDeviceptr_t deviceptr;
  // CHECK-NEXT: hipDeviceptr_t deviceptr_2;
  // CHECK-NEXT: hipTexRef texref;
  // CHECK-NEXT: hipJitOption jit_option;
  // CHECK-NEXT: hipArray_t array_;
  // CHECK-NEXT: HIP_ARRAY3D_DESCRIPTOR ARRAY3D_DESCRIPTOR;
  // CHECK-NEXT: HIP_ARRAY_DESCRIPTOR ARRAY_DESCRIPTOR;
  // CHECK-NEXT: hipIpcEventHandle_t ipcEventHandle;
  // CHECK-NEXT: hipEvent_t event_;
  // CHECK-NEXT: hipIpcMemHandle_t ipcMemHandle;
  // CHECK-NEXT: hip_Memcpy2D MEMCPY2D;
  // CHECK-NEXT: HIP_MEMCPY3D MEMCPY3D;
  // CHECK-NEXT: hipStream_t stream;
  // CHECK-NEXT: hipMipmappedArray_t mipmappedArray;
  hipDevice_t device;
  hipCtx_t context;
  hipFuncCache_t func_cache;
  hipLimit_t limit;
  hipSharedMemConfig pconfig;
  hipFunction_t function;
  hipModule_t module_;
  hipDeviceptr_t deviceptr;
  hipDeviceptr_t deviceptr_2;
  hipTexRef texref;
  hipJitOption jit_option;
  hipArray_t array_;
  HIP_ARRAY3D_DESCRIPTOR ARRAY3D_DESCRIPTOR;
  HIP_ARRAY_DESCRIPTOR ARRAY_DESCRIPTOR;
  hipIpcEventHandle_t ipcEventHandle;
  hipEvent_t event_;
  hipIpcMemHandle_t ipcMemHandle;
  hip_Memcpy2D MEMCPY2D;
  HIP_MEMCPY3D MEMCPY3D;
  hipStream_t stream;
  hipMipmappedArray_t mipmappedArray;

  // CUDA: hipError_t CUDAAPI hipInit(unsigned int Flags);
  // HIP: hipError_t hipInit(unsigned int flags);
  // CHECK: hipError_t result = hipInit(flags);
  hipError_t result = hipInit(flags);

  int driverVersion = 0;
  // CUDA: hipError_t CUDAAPI hipDriverGetVersion(int *driverVersion);
  // HIP: hipError_t hipDriverGetVersion(int* driverVersion);
  // CHECK: result = hipDriverGetVersion(&driverVersion);
  result = hipDriverGetVersion(&driverVersion);

  int ordinal = 0;
  // CUDA: hipError_t CUDAAPI hipDeviceGet(hipDevice_t *device, int ordinal);
  // HIP: hipError_t hipDeviceGet(hipDevice_t* device, int ordinal);
  // CHECK: result = hipDeviceGet(&device, ordinal);
  result = hipDeviceGet(&device, ordinal);

  int pi = 0;
  // CHECK: hipDeviceAttribute_t device_attribute = hipDeviceAttributePciBusId;
  hipDeviceAttribute_t device_attribute = hipDeviceAttributePciBusId;
  // CUDA: hipError_t CUDAAPI hipDeviceGetAttribute(int *pi, hipDeviceAttribute_t attrib, hipDevice_t dev);
  // HIP: hipError_t hipDeviceGetAttribute(int* pi, hipDeviceAttribute_t attr, int deviceId);
  // CHECK: result = hipDeviceGetAttribute(&pi, device_attribute, device);
  result = hipDeviceGetAttribute(&pi, device_attribute, device);

  int count = 0;
  // CUDA: hipError_t CUDAAPI hipGetDeviceCount(int *count);
  // HIP: hipError_t hipGetDeviceCount(int* count);
  // CHECK: result = hipGetDeviceCount(&count);
  result = hipGetDeviceCount(&count);

  // CUDA: hipError_t CUDAAPI hipDeviceTotalMem(size_t *bytes, hipDevice_t dev);
  // HIP: hipError_t hipDeviceTotalMem(size_t* bytes, hipDevice_t device);
  // CHECK: result = hipDeviceTotalMem(&bytes, device);
  // CHECK-NEXT: result = hipDeviceTotalMem(&bytes, device);
  result = hipDeviceTotalMem(&bytes, device);
  result = hipDeviceTotalMem(&bytes, device);

  int major = 0, minor = 0;
  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipDeviceComputeCapability(int *major, int *minor, hipDevice_t dev);
  // HIP: hipError_t hipDeviceComputeCapability(int* major, int* minor, hipDevice_t device);
  // CHECK: result = hipDeviceComputeCapability(&major, &minor, device);
  result = hipDeviceComputeCapability(&major, &minor, device);

  int active = 0;
  // CUDA: hipError_t CUDAAPI hipDevicePrimaryCtxGetState(hipDevice_t dev, unsigned int *flags, int *active);
  // HIP: hipError_t hipDevicePrimaryCtxGetState(hipDevice_t dev, unsigned int* flags, int* active);
  // CHECK: result = hipDevicePrimaryCtxGetState(device, &flags, &active);
  result = hipDevicePrimaryCtxGetState(device, &flags, &active);

  // CUDA: hipError_t CUDAAPI hipDevicePrimaryCtxRelease(hipDevice_t dev);
  // HIP: hipError_t hipDevicePrimaryCtxRelease(hipDevice_t dev);
  // CHECK: result = hipDevicePrimaryCtxRelease(device);
  result = hipDevicePrimaryCtxRelease(device);
#if CUDA_VERSION > 10020
  // CHECK: result = hipDevicePrimaryCtxRelease(device);
  result = hipDevicePrimaryCtxRelease(device);
#endif

  // CUDA: hipError_t CUDAAPI hipDevicePrimaryCtxReset(hipDevice_t dev);
  // HIP: hipError_t hipDevicePrimaryCtxReset(hipDevice_t dev);
  // CHECK: result = hipDevicePrimaryCtxReset(device);
  result = hipDevicePrimaryCtxReset(device);
#if CUDA_VERSION > 10020
  // CHECK: result = hipDevicePrimaryCtxReset(device);
  result = hipDevicePrimaryCtxReset(device);
#endif

  // CUDA: hipError_t CUDAAPI hipDevicePrimaryCtxRetain(hipCtx_t *pctx, hipDevice_t dev);
  // HIP: hipError_t hipDevicePrimaryCtxRetain(hipCtx_t* pctx, hipDevice_t dev);
  // CHECK: result = hipDevicePrimaryCtxRetain(&context, device);
  result = hipDevicePrimaryCtxRetain(&context, device);

  // CUDA: hipError_t CUDAAPI hipDevicePrimaryCtxSetFlags(hipDevice_t dev, unsigned int flags);
  // HIP: hipError_t hipDevicePrimaryCtxSetFlags(hipDevice_t dev, unsigned int flags);
  // CHECK: result = hipDevicePrimaryCtxSetFlags(device, flags);
  result = hipDevicePrimaryCtxSetFlags(device, flags);
#if CUDA_VERSION > 10020
  // CHECK: result = hipDevicePrimaryCtxSetFlags(device, flags);
  result = hipDevicePrimaryCtxSetFlags(device, flags);
#endif

  // CUDA: hipError_t CUDAAPI hipCtxCreate(hipCtx_t *pctx, unsigned int flags, hipDevice_t dev);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxCreate(hipCtx_t *ctx, unsigned int flags, hipDevice_t device);
  // CHECK: result = hipCtxCreate(&context, flags, device);
  // CHECK-NEXT: result = hipCtxCreate(&context, flags, device);
  result = hipCtxCreate(&context, flags, device);
  result = hipCtxCreate(&context, flags, device);

  // CUDA: hipError_t CUDAAPI hipCtxDestroy(hipCtx_t ctx);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxDestroy(hipCtx_t ctx);
  // CHECK: result = hipCtxDestroy(context);
  // CHECK-NEXT: result = hipCtxDestroy(context);
  result = hipCtxDestroy(context);
  result = hipCtxDestroy(context);

  unsigned int version = 0;
  // CUDA: hipError_t CUDAAPI hipCtxGetApiVersion(hipCtx_t ctx, unsigned int *version);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetApiVersion(hipCtx_t ctx, int* apiVersion);
  // CHECK: result = hipCtxGetApiVersion(context, &version);
  result = hipCtxGetApiVersion(context, &version);

  // CUDA: hipError_t CUDAAPI hipCtxGetCacheConfig(hipFuncCache_t *pconfig);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetCacheConfig(hipFuncCache_t* cacheConfig);
  // CHECK: result = hipCtxGetCacheConfig(&func_cache);
  result = hipCtxGetCacheConfig(&func_cache);

  // CUDA: hipError_t CUDAAPI hipCtxGetCurrent(hipCtx_t *pctx);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetCurrent(hipCtx_t* ctx);
  // CHECK: result = hipCtxGetCurrent(&context);
  result = hipCtxGetCurrent(&context);

  // CUDA: hipError_t CUDAAPI hipCtxGetDevice(hipDevice_t *device);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetDevice(hipDevice_t* device);
  // CHECK: result = hipCtxGetDevice(&device);
  result = hipCtxGetDevice(&device);

  // CUDA: hipError_t CUDAAPI hipCtxGetFlags(unsigned int *flags);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetFlags(unsigned int* flags);
  // CHECK: result = hipCtxGetFlags(&flags);
  result = hipCtxGetFlags(&flags);

  size_t pvalue = 0;
  // CUDA: hipError_t CUDAAPI hipDeviceGetLimit(size_t *pvalue, hipLimit_t limit);
  // HIP: hipError_t hipDeviceGetLimit(size_t* pValue, enum hipLimit_t limit);
  // CHECK: result = hipDeviceGetLimit(&pvalue, limit);
  result = hipDeviceGetLimit(&pvalue, limit);

  // CUDA: hipError_t CUDAAPI hipCtxGetSharedMemConfig(hipSharedMemConfig *pConfig);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetSharedMemConfig(hipSharedMemConfig* pConfig);
  // CHECK: result = hipCtxGetSharedMemConfig(&pconfig);
  result = hipCtxGetSharedMemConfig(&pconfig);

  int leastPriority = 0, greatestPriority = 0;
  // CUDA: hipError_t CUDAAPI hipDeviceGetStreamPriorityRange(int *leastPriority, int *greatestPriority);
  // HIP: hipError_t hipDeviceGetStreamPriorityRange(int* leastPriority, int* greatestPriority);
  // CHECK: result = hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority);
  result = hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority);

  // CUDA: hipError_t CUDAAPI hipCtxPopCurrent(hipCtx_t *pctx);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxPopCurrent(hipCtx_t* ctx);
  // CHECK: result = hipCtxPopCurrent(&context);
  // CHECK-NEXT: result = hipCtxPopCurrent(&context);
  result = hipCtxPopCurrent(&context);
  result = hipCtxPopCurrent(&context);

  // CUDA: hipError_t CUDAAPI hipCtxPushCurrent(hipCtx_t ctx);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxPushCurrent(hipCtx_t ctx);
  // CHECK: result = hipCtxPushCurrent(context);
  // CHECK-NEXT: result = hipCtxPushCurrent(context);
  result = hipCtxPushCurrent(context);
  result = hipCtxPushCurrent(context);

  // CUDA: hipError_t CUDAAPI hipCtxSetCacheConfig(hipFuncCache_t config);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxSetCacheConfig(hipFuncCache_t cacheConfig);
  // CHECK: result = hipCtxSetCacheConfig(func_cache);
  result = hipCtxSetCacheConfig(func_cache);

  // CUDA: hipError_t CUDAAPI hipCtxSetCurrent(hipCtx_t ctx);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxSetCurrent(hipCtx_t ctx);
  // CHECK: result = hipCtxSetCurrent(context);
  result = hipCtxSetCurrent(context);

  // CUDA: hipError_t CUDAAPI hipCtxSetSharedMemConfig(hipSharedMemConfig config);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxSetSharedMemConfig(hipSharedMemConfig config);
  // CHECK: result = hipCtxSetSharedMemConfig(pconfig);
  result = hipCtxSetSharedMemConfig(pconfig);

  // CUDA: hipError_t CUDAAPI hipCtxSynchronize(void);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxSynchronize(void);
  // CHECK: result = hipCtxSynchronize();
  result = hipCtxSynchronize();

  // CUDA: hipError_t CUDAAPI hipModuleGetFunction(hipFunction_t *hfunc, hipModule_t hmod, const char *name);
  // HIP: hipError_t hipModuleGetFunction(hipFunction_t* function, hipModule_t module, const char* kname);
  // CHECK: result = hipModuleGetFunction(&function, module_, name.c_str());
  result = hipModuleGetFunction(&function, module_, name.c_str());

  // CUDA: hipError_t CUDAAPI hipModuleGetGlobal(hipDeviceptr_t *dptr, size_t *bytes, hipModule_t hmod, const char *name);
  // HIP: hipError_t hipModuleGetGlobal(hipDeviceptr_t* dptr, size_t* bytes, hipModule_t hmod, const char* name);
  // CHECK: result = hipModuleGetGlobal(&deviceptr, &bytes, module_, name.c_str());
  // CHECK-NEXT: result = hipModuleGetGlobal(&deviceptr, &bytes, module_, name.c_str());
  result = hipModuleGetGlobal(&deviceptr, &bytes, module_, name.c_str());
  result = hipModuleGetGlobal(&deviceptr, &bytes, module_, name.c_str());

  // CUDA: hipError_t CUDAAPI hipModuleGetTexRef(hipTexRef *pTexRef, hipModule_t hmod, const char *name);
  // HIP: hipError_t hipModuleGetTexRef(textureReference** texRef, hipModule_t hmod, const char* name);
  // CHECK: result = hipModuleGetTexRef(&texref, module_, name.c_str());
  result = hipModuleGetTexRef(&texref, module_, name.c_str());

  // CUDA: hipError_t CUDAAPI hipModuleLoad(hipModule_t *module, const char *fname);
  // HIP: hipError_t hipModuleLoad(hipModule_t* module, const char* fname);
  // CHECK: result = hipModuleLoad(&module_, name.c_str());
  result = hipModuleLoad(&module_, name.c_str());

  // CUDA: hipError_t CUDAAPI hipModuleLoadData(hipModule_t *module, const void *image);
  // HIP: hipError_t hipModuleLoadData(hipModule_t* module, const void* image);
  // CHECK: result = hipModuleLoadData(&module_, image);
  result = hipModuleLoadData(&module_, image);

  unsigned int numOptions = 0;
  void* optionValues = nullptr;
  // CUDA: hipError_t CUDAAPI hipModuleLoadDataEx(hipModule_t *module, const void *image, unsigned int numOptions, hipJitOption *options, void **optionValues);
  // HIP: hipError_t hipModuleLoadDataEx(hipModule_t* module, const void* image, unsigned int numOptions, hipJitOption* options, void** optionValues);
  // CHECK: result = hipModuleLoadDataEx(&module_, image, numOptions, &jit_option, &optionValues);
  result = hipModuleLoadDataEx(&module_, image, numOptions, &jit_option, &optionValues);

  // CUDA: hipError_t CUDAAPI hipModuleUnload(hipModule_t hmod);
  // HIP: hipError_t hipModuleUnload(hipModule_t module);
  // CHECK: result = hipModuleUnload(module_);
  result = hipModuleUnload(module_);

  // CUDA: hipError_t CUDAAPI hipArray3DCreate(hipArray_t *pHandle, const HIP_ARRAY3D_DESCRIPTOR *pAllocateArray);
  // HIP: hipError_t hipArray3DCreate(hipArray** array, const HIP_ARRAY3D_DESCRIPTOR* pAllocateArray);
  // CHECK: result = hipArray3DCreate(&array_, &ARRAY3D_DESCRIPTOR);
  // CHECK-NEXT: result = hipArray3DCreate(&array_, &ARRAY3D_DESCRIPTOR);
  result = hipArray3DCreate(&array_, &ARRAY3D_DESCRIPTOR);
  result = hipArray3DCreate(&array_, &ARRAY3D_DESCRIPTOR);

  // CUDA: hipError_t CUDAAPI hipArrayCreate(hipArray_t *pHandle, const HIP_ARRAY_DESCRIPTOR *pAllocateArray);
  // HIP: hipError_t hipArrayCreate(hipArray** pHandle, const HIP_ARRAY_DESCRIPTOR* pAllocateArray);
  // CHECK: result = hipArrayCreate(&array_, &ARRAY_DESCRIPTOR);
  // CHECK: result = hipArrayCreate(&array_, &ARRAY_DESCRIPTOR);
  result = hipArrayCreate(&array_, &ARRAY_DESCRIPTOR);
  result = hipArrayCreate(&array_, &ARRAY_DESCRIPTOR);

  // CUDA: hipError_t CUDAAPI hipArrayDestroy(hipArray_t hArray);
  // HIP: hipError_t hipArrayDestroy(hipArray* array);
  // CHECK: result = hipArrayDestroy(array_);
  result = hipArrayDestroy(array_);

  std::string pciBusId;
  // CUDA: hipError_t CUDAAPI hipDeviceGetByPCIBusId(hipDevice_t *dev, const char *pciBusId);
  // HIP: hipError_t hipDeviceGetByPCIBusId(int* device, const char* pciBusId);
  // CHECK: result = hipDeviceGetByPCIBusId(&device, pciBusId.c_str());
  result = hipDeviceGetByPCIBusId(&device, pciBusId.c_str());

  int len = 0;
  char* pciBusId_ = const_cast<char*>(pciBusId.c_str());
  // CUDA: hipError_t CUDAAPI hipDeviceGetPCIBusId(char *pciBusId, int len, hipDevice_t dev);
  // HIP: hipError_t hipDeviceGetPCIBusId(char* pciBusId, int len, int device);
  // CHECK: result = hipDeviceGetPCIBusId(pciBusId_, len, device);
  result = hipDeviceGetPCIBusId(pciBusId_, len, device);

  // CUDA: hipError_t CUDAAPI hipIpcCloseMemHandle(hipDeviceptr_t dptr);
  // HIP: hipError_t hipIpcCloseMemHandle(void* devPtr);
  // CHECK: result = hipIpcCloseMemHandle(deviceptr);
  result = hipIpcCloseMemHandle(deviceptr);

  // CUDA: hipError_t CUDAAPI hipIpcGetEventHandle(hipIpcEventHandle_t *pHandle, hipEvent_t event);
  // HIP: hipError_t hipIpcGetEventHandle(hipIpcEventHandle_t* handle, hipEvent_t event);
  // CHECK: result = hipIpcGetEventHandle(&ipcEventHandle, event_);
  result = hipIpcGetEventHandle(&ipcEventHandle, event_);

  // CUDA: hipError_t CUDAAPI hipIpcGetMemHandle(hipIpcMemHandle_t *pHandle, hipDeviceptr_t dptr);
  // HIP: hipError_t hipIpcGetMemHandle(hipIpcMemHandle_t* handle, void* devPtr);
  // CHECK: result = hipIpcGetMemHandle(&ipcMemHandle, deviceptr);
  result = hipIpcGetMemHandle(&ipcMemHandle, deviceptr);

  // CUDA: hipError_t CUDAAPI hipIpcOpenEventHandle(hipEvent_t *phEvent, hipIpcEventHandle_t handle);
  // HIP: hipError_t hipIpcOpenEventHandle(hipEvent_t* event, hipIpcEventHandle_t handle);
  // CHECK: result = hipIpcOpenEventHandle(&event_, ipcEventHandle);
  result = hipIpcOpenEventHandle(&event_, ipcEventHandle);

  // CUDA: hipError_t CUDAAPI hipIpcOpenMemHandle(hipDeviceptr_t *pdptr, hipIpcMemHandle_t handle, unsigned int Flags);
  // HIP: hipError_t hipIpcOpenMemHandle(void** devPtr, hipIpcMemHandle_t handle, unsigned int flags);
  // CHECK: result = hipIpcOpenMemHandle(&deviceptr, ipcMemHandle, flags);
  result = hipIpcOpenMemHandle(&deviceptr, ipcMemHandle, flags);

  // CUDA: hipError_t CUDAAPI hipMalloc(hipDeviceptr_t *dptr, size_t bytesize);
  // HIP: hipError_t hipMalloc(void** ptr, size_t size);
  // CHECK: result = hipMalloc(&deviceptr, bytes);
  // CHECK-NEXT: result = hipMalloc(&deviceptr, bytes);
  result = hipMalloc(&deviceptr, bytes);
  result = hipMalloc(&deviceptr, bytes);

  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  ///////////// TODO: Get rid of additional attribute 'unsigned int flags' used by HIP without a default value ///////
  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

  // CUDA: hipError_t CUDAAPI hipMemAllocHost(void **pp, size_t bytesize);
  // HIP: DEPRECATED("use hipHostMalloc instead") hipError_t hipHostAlloc(void** ptr, size_t size, unsigned int flags);
  // TODO: should be hipHostAlloc(&image, bytes, 0);
  // CHECK: result = hipHostAlloc(&image, bytes);
  // CHECK-NEXT: result = hipHostAlloc(&image, bytes);
  result = hipMemAllocHost(&image, bytes);
  result = hipMemAllocHost(&image, bytes);

  // CUDA: hipError_t CUDAAPI hipMallocManaged(hipDeviceptr_t *dptr, size_t bytesize, unsigned int flags);
  // HIP: hipError_t hipMallocManaged(void** dev_ptr, size_t size, unsigned int flags __dparm(hipMemAttachGlobal));
  // CHECK: result = hipMallocManaged(&deviceptr, bytes, flags);
  result = hipMallocManaged(&deviceptr, bytes, flags);

  size_t pitch = 0, width = 0, height = 0;
  // CUDA: hipError_t CUDAAPI hipMemAllocPitch(hipDeviceptr_t *dptr, size_t *pPitch, size_t WidthInBytes, size_t Height, unsigned int ElementSizeBytes);
  // HIP: hipError_t hipMemAllocPitch(hipDeviceptr_t* dptr, size_t* pitch, size_t widthInBytes, size_t height, unsigned int elementSizeBytes);
  // CHECK: result = hipMemAllocPitch(&deviceptr, &pitch, width, height, bytes);
  // CHECK-NEXT: result = hipMemAllocPitch(&deviceptr, &pitch, width, height, bytes);
  result = hipMemAllocPitch(&deviceptr, &pitch, width, height, bytes);
  result = hipMemAllocPitch(&deviceptr, &pitch, width, height, bytes);

  // CUDA: hipError_t CUDAAPI hipMemcpyParam2D(const hip_Memcpy2D *pCopy);
  // HIP: hipError_t hipMemcpyParam2D(const hip_Memcpy2D* pCopy);
  // CHECK: result = hipMemcpyParam2D(&MEMCPY2D);
  // CHECK-NEXT: result = hipMemcpyParam2D(&MEMCPY2D);
  result = hipMemcpyParam2D(&MEMCPY2D);
  result = hipMemcpyParam2D(&MEMCPY2D);

  // CUDA: hipError_t CUDAAPI hipMemcpyParam2DAsync(const hip_Memcpy2D *pCopy, hipStream_t hStream);
  // HIP: hipError_t hipMemcpyParam2DAsync(const hip_Memcpy2D* pCopy, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemcpyParam2DAsync(&MEMCPY2D, stream);
  // CHECK-NEXT: result = hipMemcpyParam2DAsync(&MEMCPY2D, stream);
  result = hipMemcpyParam2DAsync(&MEMCPY2D, stream);
  result = hipMemcpyParam2DAsync(&MEMCPY2D, stream);

  // CUDA: hipError_t CUDAAPI hipDrvMemcpy2DUnaligned(const hip_Memcpy2D *pCopy);
  // HIP: hipError_t hipDrvMemcpy2DUnaligned(const hip_Memcpy2D* pCopy);
  // CHECK: result = hipDrvMemcpy2DUnaligned(&MEMCPY2D);
  // CHECK-NEXT: result = hipDrvMemcpy2DUnaligned(&MEMCPY2D);
  result = hipDrvMemcpy2DUnaligned(&MEMCPY2D);
  result = hipDrvMemcpy2DUnaligned(&MEMCPY2D);

  // CUDA: hipError_t CUDAAPI hipDrvMemcpy3D(const HIP_MEMCPY3D *pCopy);
  // HIP: hipError_t hipDrvMemcpy3D(const HIP_MEMCPY3D* pCopy);
  // CHECK: result = hipDrvMemcpy3D(&MEMCPY3D);
  // CHECK-NEXT: result = hipDrvMemcpy3D(&MEMCPY3D);
  result = hipDrvMemcpy3D(&MEMCPY3D);
  result = hipDrvMemcpy3D(&MEMCPY3D);

  // CUDA: hipError_t CUDAAPI hipDrvMemcpy3DAsync(const HIP_MEMCPY3D *pCopy, hipStream_t hStream);
  // HIP: hipError_t hipDrvMemcpy3DAsync(const HIP_MEMCPY3D* pCopy, hipStream_t stream);
  // CHECK: result = hipDrvMemcpy3DAsync(&MEMCPY3D, stream);
  // CHECK-NEXT: result = hipDrvMemcpy3DAsync(&MEMCPY3D, stream);
  result = hipDrvMemcpy3DAsync(&MEMCPY3D, stream);
  result = hipDrvMemcpy3DAsync(&MEMCPY3D, stream);

  void* dsthost = nullptr;
  size_t offset = 0;
  // CUDA: hipError_t CUDAAPI hipMemcpyAtoH(void *dstHost, hipArray_t srcArray, size_t srcOffset, size_t ByteCount);
  // HIP: hipError_t hipMemcpyAtoH(void* dst, hipArray* srcArray, size_t srcOffset, size_t count);
  // CHECK: result = hipMemcpyAtoH(dsthost, array_, offset, bytes);
  // CHECK-NEXT: result = hipMemcpyAtoH(dsthost, array_, offset, bytes);
  result = hipMemcpyAtoH(dsthost, array_, offset, bytes);
  result = hipMemcpyAtoH(dsthost, array_, offset, bytes);

  // CUDA: hipError_t CUDAAPI hipMemcpyDtoD(hipDeviceptr_t dstDevice, hipDeviceptr_t srcDevice, size_t ByteCount);
  // HIP: hipError_t hipMemcpyDtoD(hipDeviceptr_t dst, hipDeviceptr_t src, size_t sizeBytes);
  // CHECK: result = hipMemcpyDtoD(deviceptr, deviceptr, bytes);
  // CHECK-NEXT: result = hipMemcpyDtoD(deviceptr, deviceptr, bytes);
  result = hipMemcpyDtoD(deviceptr, deviceptr, bytes);
  result = hipMemcpyDtoD(deviceptr, deviceptr, bytes);

  // CUDA: hipError_t CUDAAPI hipMemcpyDtoDAsync(hipDeviceptr_t dstDevice, hipDeviceptr_t srcDevice, size_t ByteCount, hipStream_t hStream);
  // HIP: hipError_t hipMemcpyDtoDAsync(hipDeviceptr_t dst, hipDeviceptr_t src, size_t sizeBytes, hipStream_t stream);
  // CHECK: result = hipMemcpyDtoDAsync(deviceptr, deviceptr, bytes, stream);
  // CHECK-NEXT: result = hipMemcpyDtoDAsync(deviceptr, deviceptr, bytes, stream);
  result = hipMemcpyDtoDAsync(deviceptr, deviceptr, bytes, stream);
  result = hipMemcpyDtoDAsync(deviceptr, deviceptr, bytes, stream);

  // CUDA: hipError_t CUDAAPI hipMemcpyDtoH(void *dstHost, hipDeviceptr_t srcDevice, size_t ByteCount);
  // HIP: hipError_t hipMemcpyDtoH(void* dst, hipDeviceptr_t src, size_t sizeBytes);
  // CHECK: result = hipMemcpyDtoH(dsthost, deviceptr, bytes);
  // CHECK-NEXT: result = hipMemcpyDtoH(dsthost, deviceptr, bytes);
  result = hipMemcpyDtoH(dsthost, deviceptr, bytes);
  result = hipMemcpyDtoH(dsthost, deviceptr, bytes);

  // CUDA: hipError_t CUDAAPI hipMemcpyDtoHAsync(void *dstHost, hipDeviceptr_t srcDevice, size_t ByteCount, hipStream_t hStream);
  // HIP: hipError_t hipMemcpyDtoHAsync(void* dst, hipDeviceptr_t src, size_t sizeBytes, hipStream_t stream);
  // CHECK: result = hipMemcpyDtoHAsync(dsthost, deviceptr, bytes, stream);
  // CHECK-NEXT: result = hipMemcpyDtoHAsync(dsthost, deviceptr, bytes, stream);
  result = hipMemcpyDtoHAsync(dsthost, deviceptr, bytes, stream);
  result = hipMemcpyDtoHAsync(dsthost, deviceptr, bytes, stream);

  // CUDA: hipError_t CUDAAPI hipMemcpyHtoA(hipArray_t dstArray, size_t dstOffset, const void *srcHost, size_t ByteCount);
  // HIP: hipError_t hipMemcpyHtoA(hipArray* dstArray, size_t dstOffset, const void* srcHost, size_t count);
  // CHECK: result = hipMemcpyHtoA(array_, offset, dsthost, bytes);
  // CHECK-NEXT: result = hipMemcpyHtoA(array_, offset, dsthost, bytes);
  result = hipMemcpyHtoA(array_, offset, dsthost, bytes);
  result = hipMemcpyHtoA(array_, offset, dsthost, bytes);

  // CUDA: hipError_t CUDAAPI hipMemcpyHtoD(hipDeviceptr_t dstDevice, const void *srcHost, size_t ByteCount);
  // HIP: hipError_t hipMemcpyHtoD(hipDeviceptr_t dst, void* src, size_t sizeBytes);
  // CHECK: result = hipMemcpyHtoD(deviceptr, dsthost, bytes);
  // CHECK-NEXT: result = hipMemcpyHtoD(deviceptr, dsthost, bytes);
  result = hipMemcpyHtoD(deviceptr, dsthost, bytes);
  result = hipMemcpyHtoD(deviceptr, dsthost, bytes);

  // CUDA: hipError_t CUDAAPI hipMemcpyHtoDAsync(hipDeviceptr_t dstDevice, const void *srcHost, size_t ByteCount, hipStream_t hStream);
  // HIP: hipError_t hipMemcpyHtoDAsync(hipDeviceptr_t dst, void* src, size_t sizeBytes, hipStream_t stream);
  // CHECK: result = hipMemcpyHtoDAsync(deviceptr, dsthost, bytes, stream);
  // CHECK-NEXT: result = hipMemcpyHtoDAsync(deviceptr, dsthost, bytes, stream);
  result = hipMemcpyHtoDAsync(deviceptr, dsthost, bytes, stream);
  result = hipMemcpyHtoDAsync(deviceptr, dsthost, bytes, stream);

  // CUDA: hipError_t CUDAAPI hipFree(hipDeviceptr_t dptr);
  // HIP: hipError_t hipFree(void* ptr);
  // CHECK: result = hipFree(deviceptr);
  // CHECK-NEXT: result = hipFree(deviceptr);
  result = hipFree(deviceptr);
  result = hipFree(deviceptr);

  // CUDA: hipError_t CUDAAPI hipHostFree(void *p);
  // HIP: hipError_t hipHostFree(void* ptr);
  // CHECK: result = hipHostFree(image);
  result = hipHostFree(image);

  // CUDA: hipError_t CUDAAPI hipMemGetAddressRange(hipDeviceptr_t *pbase, size_t *psize, hipDeviceptr_t dptr);
  // HIP: hipError_t hipMemGetAddressRange(hipDeviceptr_t* pbase, size_t* psize, hipDeviceptr_t dptr);
  // CHECK: result = hipMemGetAddressRange(&deviceptr, &bytes, deviceptr_2);
  // CHECK-NEXT: result = hipMemGetAddressRange(&deviceptr, &bytes, deviceptr_2);
  result = hipMemGetAddressRange(&deviceptr, &bytes, deviceptr_2);
  result = hipMemGetAddressRange(&deviceptr, &bytes, deviceptr_2);

  // CUDA: hipError_t CUDAAPI hipMemGetInfo(size_t *free, size_t *total);
  // HIP: hipError_t hipMemGetInfo(size_t* free, size_t* total);
  // CHECK: result = hipMemGetInfo(&bytes, &bytes_2);
  // CHECK-NEXT: result = hipMemGetInfo(&bytes, &bytes_2);
  result = hipMemGetInfo(&bytes, &bytes_2);
  result = hipMemGetInfo(&bytes, &bytes_2);

  // CUDA: hipError_t CUDAAPI hipHostAlloc(void **pp, size_t bytesize, unsigned int Flags);
  // HIP: DEPRECATED("use hipHostMalloc instead") hipError_t hipHostAlloc(void** ptr, size_t size, unsigned int flags);
  // CHECK: result = hipHostAlloc(&image, bytes, flags);
  result = hipHostAlloc(&image, bytes, flags);

  // CUDA: hipError_t CUDAAPI hipHostGetDevicePointer(hipDeviceptr_t *pdptr, void *p, unsigned int Flags);
  // HIP: hipError_t hipHostGetDevicePointer(void** devPtr, void* hstPtr, unsigned int flags);
  // CHECK: result = hipHostGetDevicePointer(&deviceptr, image, flags);
  // CHECK-NEXT: result = hipHostGetDevicePointer(&deviceptr, image, flags);
  result = hipHostGetDevicePointer(&deviceptr, image, flags);
  result = hipHostGetDevicePointer(&deviceptr, image, flags);

  // CUDA: hipError_t CUDAAPI hipHostGetFlags(unsigned int *pFlags, void *p);
  // HIP: hipError_t hipHostGetFlags(&flags, image);
  // CHECK: result = hipHostGetFlags(&flags, image);
  result = hipHostGetFlags(&flags, image);

  // CUDA: hipError_t CUDAAPI hipHostRegister(void *p, size_t bytesize, unsigned int Flags);
  // HIP: hipError_t hipHostRegister(void* hostPtr, size_t sizeBytes, unsigned int flags);
  // CHECK: result = hipHostRegister(image, bytes, flags);
  // CHECK-NEXT: result = hipHostRegister(image, bytes, flags);
  result = hipHostRegister(image, bytes, flags);
  result = hipHostRegister(image, bytes, flags);

  // CUDA: hipError_t CUDAAPI hipHostUnregister(void *p);
  // HIP: hipError_t hipHostUnregister(void* hostPtr);
  // CHECK: result = hipHostUnregister(image);
  result = hipHostUnregister(image);

  unsigned short us = 0;
  // CUDA: hipError_t CUDAAPI hipMemsetD16(hipDeviceptr_t dstDevice, unsigned short us, size_t N);
  // HIP: hipError_t hipMemsetD16(hipDeviceptr_t dest, unsigned short value, size_t count);
  // CHECK: result = hipMemsetD16(deviceptr, us, bytes);
  // CHECK-NEXT: result = hipMemsetD16(deviceptr, us, bytes);
  result = hipMemsetD16(deviceptr, us, bytes);
  result = hipMemsetD16(deviceptr, us, bytes);

  // CUDA: hipError_t CUDAAPI hipMemsetD16Async(hipDeviceptr_t dstDevice, unsigned short us, size_t N, hipStream_t hStream);
  // HIP: hipError_t hipMemsetD16Async(hipDeviceptr_t dest, unsigned short value, size_t count, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemsetD16Async(deviceptr, us, bytes, stream);
  result = hipMemsetD16Async(deviceptr, us, bytes, stream);

  // CUDA: hipError_t CUDAAPI hipMemsetD32(hipDeviceptr_t dstDevice, unsigned int ui, size_t N)
  // HIP: hipError_t hipMemsetD32(hipDeviceptr_t dest, int value, size_t count);
  // CHECK: result = hipMemsetD32(deviceptr, flags, bytes);
  // CHECK-NEXT: result = hipMemsetD32(deviceptr, flags, bytes);
  result = hipMemsetD32(deviceptr, flags, bytes);
  result = hipMemsetD32(deviceptr, flags, bytes);

  // CUDA: hipError_t CUDAAPI hipMemsetD32Async(hipDeviceptr_t dstDevice, unsigned int ui, size_t N, hipStream_t hStream);
  // HIP: hipError_t hipMemsetD32Async(hipDeviceptr_t dst, int value, size_t count, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemsetD32Async(deviceptr, flags, bytes, stream);
  result = hipMemsetD32Async(deviceptr, flags, bytes, stream);

  unsigned char uc = 0;
  // CUDA: hipError_t CUDAAPI hipMemsetD8(hipDeviceptr_t dstDevice, unsigned char uc, size_t N);
  // HIP: hipError_t hipMemsetD8(hipDeviceptr_t dest, unsigned char value, size_t count);
  // CHECK: result = hipMemsetD8(deviceptr, uc, bytes);
  // CHECK-NEXT: result = hipMemsetD8(deviceptr, uc, bytes);
  result = hipMemsetD8(deviceptr, uc, bytes);
  result = hipMemsetD8(deviceptr, uc, bytes);

  // CUDA: hipError_t CUDAAPI hipMemsetD8Async(hipDeviceptr_t dstDevice, unsigned char uc, size_t N, hipStream_t hStream);
  // HIP: hipError_t hipMemsetD8Async(hipDeviceptr_t dest, unsigned char value, size_t count, hipStream_t stream __dparm(0));
  // CHECK: result = hipMemsetD8Async(deviceptr, uc, bytes, stream);
  result = hipMemsetD8Async(deviceptr, uc, bytes, stream);

  // CUDA: hipError_t CUDAAPI hipMipmappedArrayCreate(hipMipmappedArray_t *pHandle, const HIP_ARRAY3D_DESCRIPTOR *pMipmappedArrayDesc, unsigned int numMipmapLevels);
  // HIP: hipError_t hipMipmappedArrayCreate(hipMipmappedArray_t* pHandle, HIP_ARRAY3D_DESCRIPTOR* pMipmappedArrayDesc, unsigned int numMipmapLevels);
  // CHECK: result = hipMipmappedArrayCreate(&mipmappedArray, &ARRAY3D_DESCRIPTOR, flags);
  result = hipMipmappedArrayCreate(&mipmappedArray, &ARRAY3D_DESCRIPTOR, flags);

  // CUDA: hipError_t CUDAAPI hipMipmappedArrayDestroy(hipMipmappedArray_t hMipmappedArray);
  // HIP: hipError_t hipMipmappedArrayDestroy(hipMipmappedArray_t hMipmappedArray);
  // CHECK: result = hipMipmappedArrayDestroy(mipmappedArray);
  result = hipMipmappedArrayDestroy(mipmappedArray);

  // CUDA: hipError_t CUDAAPI hipMipmappedArrayGetLevel(hipArray_t *pLevelArray, hipMipmappedArray_t hMipmappedArray, unsigned int level);
  // HIP: hipError_t hipMipmappedArrayGetLevel(hipArray_t* pLevelArray, hipMipmappedArray_t hMipMappedArray, unsigned int level);
  // CHECK: result = hipMipmappedArrayGetLevel(&array_, mipmappedArray, flags);
  result = hipMipmappedArrayGetLevel(&array_, mipmappedArray, flags);

  return 0;
}
