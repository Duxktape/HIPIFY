// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>

int main() {
  printf("09. CUDA Driver API Functions synthetic test\n");

  unsigned int flags = 0;
  size_t bytes = 0;
  void* image = nullptr;
  std::string name = "function";
  // CHECK: hipDevice_t device;
  // CHECK-NEXT: hipCtx_t context;
  // CHECK-NEXT: hipFuncCache_t func_cache;
  // CHECK-NEXT: hipLimit_t limit;
  // CHECK-NEXT: hipSharedMemConfig pconfig;
  // CHECK-NEXT: hipFunction_t function;
  // CHECK-NEXT: hipModule_t module_;
  // CHECK-NEXT: hipDeviceptr_t deviceptr;
  // CHECK-NEXT: hipTexRef texref;
  // CHECK-NEXT: hipJitOption jit_option;
  hipDevice_t device;
  hipCtx_t context;
  hipFuncCache_t func_cache;
  hipLimit_t limit;
  hipSharedMemConfig pconfig;
  hipFunction_t function;
  hipModule_t module_;
  hipDeviceptr_t deviceptr;
  hipTexRef texref;
  hipJitOption jit_option;

  // CUDA: hipError_t CUDAAPI hipInit(unsigned int Flags);
  // HIP: hipError_t hipInit(unsigned int flags);
  // CHECK: hipError_t result = hipInit(flags);
  hipError_t result = hipInit(flags);

  int driverVersion = 0;
  // CUDA: hipError_t CUDAAPI hipDriverGetVersion(int *driverVersion);
  // HIP: hipError_t hipDriverGetVersion(int* driverVersion);
  // CHECK: result = hipDriverGetVersion(&driverVersion);
  result = hipDriverGetVersion(&driverVersion);

  int ordinal = 0;
  // CUDA: hipError_t CUDAAPI hipDeviceGet(hipDevice_t *device, int ordinal);
  // HIP: hipError_t hipDeviceGet(hipDevice_t* device, int ordinal);
  // CHECK: result = hipDeviceGet(&device, ordinal);
  result = hipDeviceGet(&device, ordinal);

  int pi = 0;
  // CHECK: hipDeviceAttribute_t device_attribute = hipDeviceAttributePciBusId;
  hipDeviceAttribute_t device_attribute = hipDeviceAttributePciBusId;
  // CUDA: hipError_t CUDAAPI hipDeviceGetAttribute(int *pi, hipDeviceAttribute_t attrib, hipDevice_t dev);
  // HIP: hipError_t hipDeviceGetAttribute(int* pi, hipDeviceAttribute_t attr, int deviceId);
  // CHECK: result = hipDeviceGetAttribute(&pi, device_attribute, device);
  result = hipDeviceGetAttribute(&pi, device_attribute, device);

  int count = 0;
  // CUDA: hipError_t CUDAAPI hipGetDeviceCount(int *count);
  // HIP: hipError_t hipGetDeviceCount(int* count);
  // CHECK: result = hipGetDeviceCount(&count);
  result = hipGetDeviceCount(&count);

  // CUDA: hipError_t CUDAAPI hipDeviceTotalMem(size_t *bytes, hipDevice_t dev);
  // HIP: hipError_t hipDeviceTotalMem(size_t* bytes, hipDevice_t device);
  // CHECK: result = hipDeviceTotalMem(&bytes, device);
  // CHECK-NEXT: result = hipDeviceTotalMem(&bytes, device);
  result = hipDeviceTotalMem(&bytes, device);
  result = hipDeviceTotalMem(&bytes, device);

  int major = 0, minor = 0;
  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipDeviceComputeCapability(int *major, int *minor, hipDevice_t dev);
  // HIP: hipError_t hipDeviceComputeCapability(int* major, int* minor, hipDevice_t device);
  // CHECK: result = hipDeviceComputeCapability(&major, &minor, device);
  result = hipDeviceComputeCapability(&major, &minor, device);

  int active = 0;
  // CUDA: hipError_t CUDAAPI hipDevicePrimaryCtxGetState(hipDevice_t dev, unsigned int *flags, int *active);
  // HIP: hipError_t hipDevicePrimaryCtxGetState(hipDevice_t dev, unsigned int* flags, int* active);
  // CHECK: result = hipDevicePrimaryCtxGetState(device, &flags, &active);
  result = hipDevicePrimaryCtxGetState(device, &flags, &active);

  // CUDA: hipError_t CUDAAPI hipDevicePrimaryCtxRelease(hipDevice_t dev);
  // HIP: hipError_t hipDevicePrimaryCtxRelease(hipDevice_t dev);
  // CHECK: result = hipDevicePrimaryCtxRelease(device);
  // CHECK-NEXT: result = hipDevicePrimaryCtxRelease(device);
  result = hipDevicePrimaryCtxRelease(device);
  result = hipDevicePrimaryCtxRelease(device);

  // CUDA: hipError_t CUDAAPI hipDevicePrimaryCtxReset(hipDevice_t dev);
  // HIP: hipError_t hipDevicePrimaryCtxReset(hipDevice_t dev);
  // CHECK: result = hipDevicePrimaryCtxReset(device);
  // CHECK-NEXT: result = hipDevicePrimaryCtxReset(device);
  result = hipDevicePrimaryCtxReset(device);
  result = hipDevicePrimaryCtxReset(device);

  // CUDA: hipError_t CUDAAPI hipDevicePrimaryCtxRetain(hipCtx_t *pctx, hipDevice_t dev);
  // HIP: hipError_t hipDevicePrimaryCtxRetain(hipCtx_t* pctx, hipDevice_t dev);
  // CHECK: result = hipDevicePrimaryCtxRetain(&context, device);
  result = hipDevicePrimaryCtxRetain(&context, device);

  // CUDA: hipError_t CUDAAPI hipDevicePrimaryCtxSetFlags(hipDevice_t dev, unsigned int flags);
  // HIP: hipError_t hipDevicePrimaryCtxSetFlags(hipDevice_t dev, unsigned int flags);
  // CHECK: result = hipDevicePrimaryCtxSetFlags(device, flags);
  // CHECK-NEXT: result = hipDevicePrimaryCtxSetFlags(device, flags);
  result = hipDevicePrimaryCtxSetFlags(device, flags);
  result = hipDevicePrimaryCtxSetFlags(device, flags);

  // CUDA: hipError_t CUDAAPI hipCtxCreate(hipCtx_t *pctx, unsigned int flags, hipDevice_t dev);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxCreate(hipCtx_t *ctx, unsigned int flags, hipDevice_t device);
  // CHECK: result = hipCtxCreate(&context, flags, device);
  // CHECK-NEXT: result = hipCtxCreate(&context, flags, device);
  result = hipCtxCreate(&context, flags, device);
  result = hipCtxCreate(&context, flags, device);

  // CUDA: hipError_t CUDAAPI hipCtxDestroy(hipCtx_t ctx);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxDestroy(hipCtx_t ctx);
  // CHECK: result = hipCtxDestroy(context);
  // CHECK-NEXT: result = hipCtxDestroy(context);
  result = hipCtxDestroy(context);
  result = hipCtxDestroy(context);

  unsigned int version = 0;
  // CUDA: hipError_t CUDAAPI hipCtxGetApiVersion(hipCtx_t ctx, unsigned int *version);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetApiVersion(hipCtx_t ctx, int* apiVersion);
  // CHECK: result = hipCtxGetApiVersion(context, &version);
  result = hipCtxGetApiVersion(context, &version);

  // CUDA: hipError_t CUDAAPI hipCtxGetCacheConfig(hipFuncCache_t *pconfig);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetCacheConfig(hipFuncCache_t* cacheConfig);
  // CHECK: result = hipCtxGetCacheConfig(&func_cache);
  result = hipCtxGetCacheConfig(&func_cache);

  // CUDA: hipError_t CUDAAPI hipCtxGetCurrent(hipCtx_t *pctx);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetCurrent(hipCtx_t* ctx);
  // CHECK: result = hipCtxGetCurrent(&context);
  result = hipCtxGetCurrent(&context);

  // CUDA: hipError_t CUDAAPI hipCtxGetDevice(hipDevice_t *device);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetDevice(hipDevice_t* device);
  // CHECK: result = hipCtxGetDevice(&device);
  result = hipCtxGetDevice(&device);

  // CUDA: hipError_t CUDAAPI hipCtxGetFlags(unsigned int *flags);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetFlags(unsigned int* flags);
  // CHECK: result = hipCtxGetFlags(&flags);
  result = hipCtxGetFlags(&flags);

  size_t pvalue = 0;
  // CUDA: hipError_t CUDAAPI hipDeviceGetLimit(size_t *pvalue, hipLimit_t limit);
  // HIP: hipError_t hipDeviceGetLimit(size_t* pValue, enum hipLimit_t limit);
  // CHECK: result = hipDeviceGetLimit(&pvalue, limit);
  result = hipDeviceGetLimit(&pvalue, limit);

  // CUDA: hipError_t CUDAAPI hipCtxGetSharedMemConfig(hipSharedMemConfig *pConfig);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetSharedMemConfig(hipSharedMemConfig* pConfig);
  // CHECK: result = hipCtxGetSharedMemConfig(&pconfig);
  result = hipCtxGetSharedMemConfig(&pconfig);

  int leastPriority = 0, greatestPriority = 0;
  // CUDA: hipError_t CUDAAPI hipDeviceGetStreamPriorityRange(int *leastPriority, int *greatestPriority);
  // HIP: hipError_t hipDeviceGetStreamPriorityRange(int* leastPriority, int* greatestPriority);
  // CHECK: result = hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority);
  result = hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority);

  // CUDA: hipError_t CUDAAPI hipCtxPopCurrent(hipCtx_t *pctx);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxPopCurrent(hipCtx_t* ctx);
  // CHECK: result = hipCtxPopCurrent(&context);
  // CHECK-NEXT: result = hipCtxPopCurrent(&context);
  result = hipCtxPopCurrent(&context);
  result = hipCtxPopCurrent(&context);

  // CUDA: hipError_t CUDAAPI hipCtxPushCurrent(hipCtx_t ctx);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxPushCurrent(hipCtx_t ctx);
  // CHECK: result = hipCtxPushCurrent(context);
  // CHECK-NEXT: result = hipCtxPushCurrent(context);
  result = hipCtxPushCurrent(context);
  result = hipCtxPushCurrent(context);

  // CUDA: hipError_t CUDAAPI hipCtxSetCacheConfig(hipFuncCache_t config);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxSetCacheConfig(hipFuncCache_t cacheConfig);
  // CHECK: result = hipCtxSetCacheConfig(func_cache);
  result = hipCtxSetCacheConfig(func_cache);

  // CUDA: hipError_t CUDAAPI hipCtxSetCurrent(hipCtx_t ctx);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxSetCurrent(hipCtx_t ctx);
  // CHECK: result = hipCtxSetCurrent(context);
  result = hipCtxSetCurrent(context);

  // CUDA: hipError_t CUDAAPI hipCtxSetSharedMemConfig(hipSharedMemConfig config);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxSetSharedMemConfig(hipSharedMemConfig config);
  // CHECK: result = hipCtxSetSharedMemConfig(pconfig);
  result = hipCtxSetSharedMemConfig(pconfig);

  // CUDA: hipError_t CUDAAPI hipCtxSynchronize(void);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxSynchronize(void);
  // CHECK: result = hipCtxSynchronize();
  result = hipCtxSynchronize();

  // CUDA: hipError_t CUDAAPI hipModuleGetFunction(hipFunction_t *hfunc, hipModule_t hmod, const char *name);
  // HIP: hipError_t hipModuleGetFunction(hipFunction_t* function, hipModule_t module, const char* kname);
  // CHECK: result = hipModuleGetFunction(&function, module_, name.c_str());
  result = hipModuleGetFunction(&function, module_, name.c_str());

  // CUDA: hipError_t CUDAAPI hipModuleGetGlobal(hipDeviceptr_t *dptr, size_t *bytes, hipModule_t hmod, const char *name);
  // HIP: hipError_t hipModuleGetGlobal(hipDeviceptr_t* dptr, size_t* bytes, hipModule_t hmod, const char* name);
  // CHECK: result = hipModuleGetGlobal(&deviceptr, &bytes, module_, name.c_str());
  // CHECK: result = hipModuleGetGlobal(&deviceptr, &bytes, module_, name.c_str());
  result = hipModuleGetGlobal(&deviceptr, &bytes, module_, name.c_str());
  result = hipModuleGetGlobal(&deviceptr, &bytes, module_, name.c_str());

  // CUDA: hipError_t CUDAAPI hipModuleGetTexRef(hipTexRef *pTexRef, hipModule_t hmod, const char *name);
  // HIP: hipError_t hipModuleGetTexRef(textureReference** texRef, hipModule_t hmod, const char* name);
  // CHECK: result = hipModuleGetTexRef(&texref, module_, name.c_str());
  result = hipModuleGetTexRef(&texref, module_, name.c_str());

  // CUDA: hipError_t CUDAAPI hipModuleLoad(hipModule_t *module, const char *fname);
  // HIP: hipError_t hipModuleLoad(hipModule_t* module, const char* fname);
  // CHECK: result = hipModuleLoad(&module_, name.c_str());
  result = hipModuleLoad(&module_, name.c_str());

  // CUDA: hipError_t CUDAAPI hipModuleLoadData(hipModule_t *module, const void *image);
  // HIP: hipError_t hipModuleLoadData(hipModule_t* module, const void* image);
  // CHECK: result = hipModuleLoadData(&module_, image);
  result = hipModuleLoadData(&module_, image);

  unsigned int numOptions = 0;
  void* optionValues = nullptr;
  // CUDA: hipError_t CUDAAPI hipModuleLoadDataEx(hipModule_t *module, const void *image, unsigned int numOptions, hipJitOption *options, void **optionValues);
  // HIP: hipError_t hipModuleLoadDataEx(hipModule_t* module, const void* image, unsigned int numOptions, hipJitOption* options, void** optionValues);
  // CHECK: result = hipModuleLoadDataEx(&module_, image, numOptions, &jit_option, &optionValues);
  result = hipModuleLoadDataEx(&module_, image, numOptions, &jit_option, &optionValues);

  // CUDA: hipError_t CUDAAPI hipModuleUnload(hipModule_t hmod);
  // HIP: hipError_t hipModuleUnload(hipModule_t module);
  // CHECK: result = hipModuleUnload(module_);
  result = hipModuleUnload(module_);

  return 0;
}
