// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int main() {
  printf("09. CUDA Driver API Functions synthetic test\n");

  unsigned int flags = 0;
  // CHECK: hipDevice_t device;
  // CHECK-NEXT: hipCtx_t context;
  // CHECK-NEXT: hipFuncCache_t func_cache;
  hipDevice_t device;
  hipCtx_t context;
  hipFuncCache_t func_cache;

  // CUDA: hipError_t CUDAAPI hipInit(unsigned int Flags);
  // HIP: hipError_t hipInit(unsigned int flags);
  // CHECK: hipError_t result = hipInit(flags);
  hipError_t result = hipInit(flags);

  int driverVersion = 0;
  // CUDA: hipError_t CUDAAPI hipDriverGetVersion(int *driverVersion);
  // HIP: hipError_t hipDriverGetVersion(int* driverVersion);
  // CHECK: result = hipDriverGetVersion(&driverVersion);
  result = hipDriverGetVersion(&driverVersion);

  int ordinal = 0;
  // CUDA: hipError_t CUDAAPI hipDeviceGet(hipDevice_t *device, int ordinal);
  // HIP: hipError_t hipDeviceGet(hipDevice_t* device, int ordinal);
  // CHECK: result = hipDeviceGet(&device, ordinal);
  result = hipDeviceGet(&device, ordinal);

  int pi = 0;
  // CHECK: hipDeviceAttribute_t device_attribute = hipDeviceAttributePciBusId;
  hipDeviceAttribute_t device_attribute = hipDeviceAttributePciBusId;
  // CUDA: hipError_t CUDAAPI hipDeviceGetAttribute(int *pi, hipDeviceAttribute_t attrib, hipDevice_t dev);
  // HIP: hipError_t hipDeviceGetAttribute(int* pi, hipDeviceAttribute_t attr, int deviceId);
  // CHECK: result = hipDeviceGetAttribute(&pi, device_attribute, device);
  result = hipDeviceGetAttribute(&pi, device_attribute, device);

  int count = 0;
  // CUDA: hipError_t CUDAAPI hipGetDeviceCount(int *count);
  // HIP: hipError_t hipGetDeviceCount(int* count);
  // CHECK: result = hipGetDeviceCount(&count);
  result = hipGetDeviceCount(&count);

  size_t bytes = 0;
  // CUDA: hipError_t CUDAAPI hipDeviceTotalMem(size_t *bytes, hipDevice_t dev);
  // HIP: hipError_t hipDeviceTotalMem(size_t* bytes, hipDevice_t device);
  // CHECK: result = hipDeviceTotalMem(&bytes, device);
  // CHECK-NEXT: result = hipDeviceTotalMem(&bytes, device);
  result = hipDeviceTotalMem(&bytes, device);
  result = hipDeviceTotalMem(&bytes, device);

  int major = 0, minor = 0;
  // CUDA: __CUDA_DEPRECATED hipError_t CUDAAPI hipDeviceComputeCapability(int *major, int *minor, hipDevice_t dev);
  // HIP: hipError_t hipDeviceComputeCapability(int* major, int* minor, hipDevice_t device);
  // CHECK: result = hipDeviceComputeCapability(&major, &minor, device);
  result = hipDeviceComputeCapability(&major, &minor, device);

  int active = 0;
  // CUDA: hipError_t CUDAAPI hipDevicePrimaryCtxGetState(hipDevice_t dev, unsigned int *flags, int *active);
  // HIP: hipError_t hipDevicePrimaryCtxGetState(hipDevice_t dev, unsigned int* flags, int* active);
  // CHECK: result = hipDevicePrimaryCtxGetState(device, &flags, &active);
  result = hipDevicePrimaryCtxGetState(device, &flags, &active);

  // CUDA: hipError_t CUDAAPI hipDevicePrimaryCtxRelease(hipDevice_t dev);
  // HIP: hipError_t hipDevicePrimaryCtxRelease(hipDevice_t dev);
  // CHECK: result = hipDevicePrimaryCtxRelease(device);
  // CHECK-NEXT: result = hipDevicePrimaryCtxRelease(device);
  result = hipDevicePrimaryCtxRelease(device);
  result = hipDevicePrimaryCtxRelease(device);

  // CUDA: hipError_t CUDAAPI hipDevicePrimaryCtxReset(hipDevice_t dev);
  // HIP: hipError_t hipDevicePrimaryCtxReset(hipDevice_t dev);
  // CHECK: result = hipDevicePrimaryCtxReset(device);
  // CHECK-NEXT: result = hipDevicePrimaryCtxReset(device);
  result = hipDevicePrimaryCtxReset(device);
  result = hipDevicePrimaryCtxReset(device);

  // CUDA: hipError_t CUDAAPI hipDevicePrimaryCtxRetain(hipCtx_t *pctx, hipDevice_t dev);
  // HIP: hipError_t hipDevicePrimaryCtxRetain(hipCtx_t* pctx, hipDevice_t dev);
  // CHECK: result = hipDevicePrimaryCtxRetain(&context, device);
  result = hipDevicePrimaryCtxRetain(&context, device);

  // CUDA: hipError_t CUDAAPI hipDevicePrimaryCtxSetFlags(hipDevice_t dev, unsigned int flags);
  // HIP: hipError_t hipDevicePrimaryCtxSetFlags(hipDevice_t dev, unsigned int flags);
  // CHECK: result = hipDevicePrimaryCtxSetFlags(device, flags);
  // CHECK-NEXT: result = hipDevicePrimaryCtxSetFlags(device, flags);
  result = hipDevicePrimaryCtxSetFlags(device, flags);
  result = hipDevicePrimaryCtxSetFlags(device, flags);

  // CUDA: hipError_t CUDAAPI hipCtxCreate(hipCtx_t *pctx, unsigned int flags, hipDevice_t dev);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxCreate(hipCtx_t *ctx, unsigned int flags, hipDevice_t device);
  // CHECK: result = hipCtxCreate(&context, flags, device);
  // CHECK-NEXT: result = hipCtxCreate(&context, flags, device);
  result = hipCtxCreate(&context, flags, device);
  result = hipCtxCreate(&context, flags, device);

  // CUDA: hipError_t CUDAAPI hipCtxDestroy(hipCtx_t ctx);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxDestroy(hipCtx_t ctx);
  // CHECK: result = hipCtxDestroy(context);
  // CHECK-NEXT: result = hipCtxDestroy(context);
  result = hipCtxDestroy(context);
  result = hipCtxDestroy(context);

  unsigned int version = 0;
  // CUDA: hipError_t CUDAAPI hipCtxGetApiVersion(hipCtx_t ctx, unsigned int *version);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetApiVersion(hipCtx_t ctx, int* apiVersion);
  // CHECK: result = hipCtxGetApiVersion(context, &version);
  result = hipCtxGetApiVersion(context, &version);

  // CUDA: hipError_t CUDAAPI hipCtxGetCacheConfig(hipFuncCache_t *pconfig);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetCacheConfig(hipFuncCache_t* cacheConfig);
  // CHECK: result = hipCtxGetCacheConfig(&func_cache);
  result = hipCtxGetCacheConfig(&func_cache);

  // CUDA: hipError_t CUDAAPI hipCtxGetCurrent(hipCtx_t *pctx);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetCurrent(hipCtx_t* ctx);
  // CHECK: result = hipCtxGetCurrent(&context);
  result = hipCtxGetCurrent(&context);

  // CUDA: hipError_t CUDAAPI hipCtxGetDevice(hipDevice_t *device);
  // HIP: DEPRECATED(DEPRECATED_MSG) hipError_t hipCtxGetDevice(hipDevice_t* device);
  // CHECK: result = hipCtxGetDevice(&device);
  result = hipCtxGetDevice(&device);

  return 0;
}
